#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ static inline int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}
//

__global__ void mandelKernel(int *d_res, double lowerX, double lowerY, double stepX, double stepY, int resX, int resY, int pitch, int maxIterations)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    
    int myX, myY;
    myX = blockIdx.x*blockDim.x + threadIdx.x;
    myY = blockIdx.y*blockDim.y + threadIdx.y;
    if (myX>=resX||myY>=resY) return;

    float x, y;
    x = lowerX + myX * stepX;
    y = lowerY + myY * stepY;


    int color = mandel (x , y, maxIterations) ; 
    d_res[myY * resX + myX] = color;
    
}
//
// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int blocksX, blocksY;
    blocksX = (int) ceil (resX / 16);
    blocksY= (int)ceil(resY/ 4);

    dim3 block (16, 4);
    dim3 grid ( blocksX , blocksY );

    int pitch;
    int *h_res;
    int *d_res;

    hipMallocPitch((void **)&d_res, (size_t *)&pitch, sizeof(int)*resX, resY);
    hipHostAlloc(&h_res, resX*resY*sizeof(int), hipHostMallocMapped);
    mandelKernel <<<grid, block>>> ( d_res ,lowerX, lowerY, stepX ,stepY, resX, resY, pitch,maxIterations);

    hipDeviceSynchronize();
    hipMemcpy(h_res, d_res, resY * resX * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0 ; i<resY ; i++){
      for(int j=0 ; j<resX ; j++){
        img[i*resX + j] = h_res[i*resX+j];
      }
    }
    hipHostFree(h_res);
    hipFree(d_res);
}

