#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ static inline int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

//how many pixel does a thread compute

static const int thr_blk_x = 4;
static const int thr_blk_y = 4;
static const int block_side = 16;

__global__ void mandelKernel(int *d_res, float lowerX, float lowerY, float stepX, float  stepY, int resX, int resY, int pitch, int maxIterations)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    
    int myX, myY;
    myX = (blockIdx.x*blockDim.x + threadIdx.x)*thr_blk_x;
    myY = (blockIdx.y*blockDim.y + threadIdx.y)*thr_blk_y;

    int i,j;

    for(i = myX ; i<myX+thr_blk_x ; i++){
      for(j = myY ; j<myY+thr_blk_y ; j++){
        if (i>=resX || j>=resY) continue;
        float x, y;
        x = lowerX + i * stepX;
        y = lowerY +j * stepY;
        
        int color = mandel (x, y, maxIterations) ; 
        d_res[j * resX + i] = color;
      }
    }


}
//
// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int blocksX, blocksY;
    blocksX = (int) ceil (resX*1.0 / (block_side*thr_blk_x));
    blocksY= (int)ceil(resY*1.0/ (block_side*thr_blk_y));

    dim3 block (block_side, block_side);
    dim3 grid (blocksX , blocksY);

    size_t pitch;
    int *h_res;
    int *d_res;

    hipMallocPitch((void **)&d_res, (size_t *)&pitch, sizeof(int)*resX, resY);
    hipHostAlloc(&h_res, resX*resY*sizeof(int), hipHostMallocMapped);
    mandelKernel <<<grid, block>>> ( d_res ,lowerX, lowerY, stepX ,stepY, resX, resY, pitch,maxIterations);

    hipDeviceSynchronize();
    hipMemcpy(h_res, d_res, resY * resX * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0 ; i<resY ; i++){
      for(int j=0 ; j<resX ; j++){
        img[i*resX + j] = h_res[i*resX+j];
      }
    }
    hipHostFree(h_res);
    hipFree(d_res);
}